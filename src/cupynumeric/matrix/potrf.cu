/* Copyright 2024 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cupynumeric/matrix/potrf.h"
#include "cupynumeric/matrix/potrf_template.inl"

#include "cupynumeric/cuda_help.h"

namespace cupynumeric {

using namespace legate;

template <typename PotrfBufferSize, typename Potrf, typename VAL>
static inline void potrf_template(
  PotrfBufferSize potrfBufferSize, Potrf potrf, VAL* array, int32_t m, int32_t n)
{
  auto uplo = HIPBLAS_FILL_MODE_LOWER;

  auto context = get_cusolver();
  auto stream  = get_cached_stream();
  CHECK_CUSOLVER(hipsolverSetStream(context, stream));

  int32_t bufferSize;
  CHECK_CUSOLVER(potrfBufferSize(context, uplo, n, array, m, &bufferSize));

  auto buffer = create_buffer<VAL>(bufferSize, Memory::Kind::GPU_FB_MEM);
  auto info   = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  CHECK_CUSOLVER(potrf(context, uplo, n, array, m, buffer.ptr(0), bufferSize, info.ptr(0)));

  // TODO: We need a deferred exception to avoid this synchronization
  CUPYNUMERIC_CHECK_CUDA(hipStreamSynchronize(stream));
  CUPYNUMERIC_CHECK_CUDA_STREAM(stream);

  if (info[0] != 0) {
    throw legate::TaskException("Matrix is not positive definite");
  }
}

template <>
void PotrfImplBody<VariantKind::GPU, Type::Code::FLOAT32>::operator()(float* array,
                                                                      int32_t m,
                                                                      int32_t n)
{
  potrf_template(hipsolverDnSpotrf_bufferSize, hipsolverDnSpotrf, array, m, n);
}

template <>
void PotrfImplBody<VariantKind::GPU, Type::Code::FLOAT64>::operator()(double* array,
                                                                      int32_t m,
                                                                      int32_t n)
{
  potrf_template(hipsolverDnDpotrf_bufferSize, hipsolverDnDpotrf, array, m, n);
}

template <>
void PotrfImplBody<VariantKind::GPU, Type::Code::COMPLEX64>::operator()(complex<float>* array,
                                                                        int32_t m,
                                                                        int32_t n)
{
  potrf_template(
    hipsolverDnCpotrf_bufferSize, hipsolverDnCpotrf, reinterpret_cast<hipComplex*>(array), m, n);
}

template <>
void PotrfImplBody<VariantKind::GPU, Type::Code::COMPLEX128>::operator()(complex<double>* array,
                                                                         int32_t m,
                                                                         int32_t n)
{
  potrf_template(
    hipsolverDnZpotrf_bufferSize, hipsolverDnZpotrf, reinterpret_cast<hipDoubleComplex*>(array), m, n);
}

/*static*/ void PotrfTask::gpu_variant(TaskContext context)
{
  potrf_template<VariantKind::GPU>(context);
}

}  // namespace cupynumeric
