#include "hip/hip_runtime.h"
/* Copyright 2024 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cupynumeric/matrix/syev.h"
#include "cupynumeric/matrix/syev_template.inl"
#include "cupynumeric/utilities/thrust_util.h"

#include <thrust/execution_policy.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include "cupynumeric/cuda_help.h"
#include <vector>

namespace cupynumeric {

using namespace legate;

template <typename VAL_COMPLEX>
struct removeDiagImag {
  VAL_COMPLEX* a_inout_;
  const int64_t m_;

  removeDiagImag(VAL_COMPLEX* a_inout, int64_t m) : a_inout_(a_inout), m_(m) {}

  __CUDA_HD__ void operator()(const int64_t& idx) const
  {
    int64_t idx_diag     = idx * m_ + idx % m_;
    a_inout_[idx_diag].y = 0.0;
  }
};

template <typename VAL_COMPLEX>
void remove_diag_imag(VAL_COMPLEX* a_inout, int64_t m, int64_t num_batches = 1)
{
  auto stream = get_cached_stream();
  thrust::for_each(DEFAULT_POLICY.on(stream),
                   thrust::make_counting_iterator<int64_t>(0),
                   thrust::make_counting_iterator<int64_t>(m * num_batches),
                   removeDiagImag<VAL_COMPLEX>(a_inout, m));
}

template <typename VAL, typename DataType>
static inline void syev_batched_template(DataType valTypeR,
                                         DataType valTypeA,
                                         bool uplo_l,
                                         int64_t m,
                                         const void* a,
                                         void* ew,
                                         void* ev,
                                         int64_t num_batches)
{
  auto handle       = get_cusolver();
  auto stream       = get_cached_stream();
  auto syev_handles = get_cusolver_extra_symbols();

  assert(syev_handles->has_syev_batched);

  bool compute_evs = ev != nullptr;

  auto a_copy = create_buffer<VAL>(compute_evs ? 0 : num_batches * m * m, Memory::Kind::GPU_FB_MEM);
  void* a_copy_ptr = compute_evs ? ev : reinterpret_cast<void*>(a_copy.ptr(0));

  CUPYNUMERIC_CHECK_CUDA(hipMemcpyAsync(
    a_copy_ptr, a, num_batches * m * m * sizeof(VAL), hipMemcpyDeviceToDevice, stream));

  if constexpr (std::is_same_v<VAL, complex<float>>) {
    remove_diag_imag(reinterpret_cast<hipComplex*>(a_copy_ptr), m, num_batches);
  } else if constexpr (std::is_same_v<VAL, complex<double>>) {
    remove_diag_imag(reinterpret_cast<hipDoubleComplex*>(a_copy_ptr), m, num_batches);
  }

  CHECK_CUSOLVER(hipsolverSetStream(handle, stream));

  size_t lwork_device, lwork_host;
  CHECK_CUSOLVER(syev_handles->cusolver_syev_batched_bufferSize(
    handle,
    nullptr,
    compute_evs ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR,
    uplo_l ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER,
    m,
    valTypeA,
    a_copy_ptr,
    m,
    valTypeR,
    ew,
    valTypeA,
    &lwork_device,
    &lwork_host,
    num_batches));

  auto buffer = create_buffer<char>(lwork_device, Memory::Kind::GPU_FB_MEM);
  std::vector<char> buffer_host(std::max(1ul, lwork_host));
  auto info = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  CHECK_CUSOLVER(syev_handles->cusolver_syev_batched(
    handle,
    nullptr,
    compute_evs ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR,
    uplo_l ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER,
    m,
    valTypeA,
    a_copy_ptr,
    m,
    valTypeR,
    ew,
    valTypeA,
    buffer.ptr(0),
    lwork_device,
    buffer_host.data(),
    lwork_host,
    info.ptr(0),
    num_batches));

  CUPYNUMERIC_CHECK_CUDA_STREAM(stream);

  if (info[0] != 0) {
    throw legate::TaskException(SyevTask::ERROR_MESSAGE);
  }
}

template <typename VAL, typename DataType>
static inline void syevd_template(
  DataType valTypeR, DataType valTypeA, bool uplo_l, int64_t m, const void* a, void* ew, void* ev)
{
  auto handle = get_cusolver();
  auto stream = get_cached_stream();

  bool compute_evs = ev != nullptr;

  auto a_copy      = create_buffer<VAL>(compute_evs ? 0 : m * m, Memory::Kind::GPU_FB_MEM);
  void* a_copy_ptr = compute_evs ? ev : reinterpret_cast<void*>(a_copy.ptr(0));

  CUPYNUMERIC_CHECK_CUDA(
    hipMemcpyAsync(a_copy_ptr, a, m * m * sizeof(VAL), hipMemcpyDeviceToDevice, stream));

  if constexpr (std::is_same_v<VAL, complex<float>>) {
    remove_diag_imag(reinterpret_cast<hipComplex*>(a_copy_ptr), m);
  } else if constexpr (std::is_same_v<VAL, complex<double>>) {
    remove_diag_imag(reinterpret_cast<hipDoubleComplex*>(a_copy_ptr), m);
  }

  CHECK_CUSOLVER(hipsolverSetStream(handle, stream));

  size_t lwork_device, lwork_host;
  CHECK_CUSOLVER(
    cusolverDnXsyevd_bufferSize(handle,
                                nullptr,
                                compute_evs ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR,
                                uplo_l ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER,
                                m,
                                valTypeA,
                                a_copy_ptr,
                                m,
                                valTypeR,
                                ew,
                                valTypeA,
                                &lwork_device,
                                &lwork_host));

  auto buffer = create_buffer<char>(lwork_device, Memory::Kind::GPU_FB_MEM);
  std::vector<char> buffer_host(std::max(1ul, lwork_host));
  auto info = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  CHECK_CUSOLVER(
    cusolverDnXsyevd(handle,
                     nullptr,
                     compute_evs ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR,
                     uplo_l ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER,
                     m,
                     valTypeA,
                     a_copy_ptr,
                     m,
                     valTypeR,
                     ew,
                     valTypeA,
                     buffer.ptr(0),
                     lwork_device,
                     buffer_host.data(),
                     lwork_host,
                     info.ptr(0)));

  CUPYNUMERIC_CHECK_CUDA_STREAM(stream);

  if (info[0] != 0) {
    throw legate::TaskException(SyevTask::ERROR_MESSAGE);
  }
}

template <>
struct SyevImplBody<VariantKind::GPU, Type::Code::FLOAT32> {
  void operator()(bool uplo_l,
                  int64_t m,
                  int64_t num_batches,
                  int64_t batch_stride_ew,
                  int64_t batch_stride_ev,
                  const float* a,
                  float* ew,
                  float* ev)
  {
    bool compute_evs = ev != nullptr;

    if (num_batches > 1 && get_cusolver_extra_symbols()->has_syev_batched) {
      syev_batched_template<float>(
        HIP_R_32F, HIP_R_32F, uplo_l, m, a, ew, compute_evs ? ev : nullptr, num_batches);
    } else {
      for (int64_t batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        syevd_template<float>(HIP_R_32F,
                              HIP_R_32F,
                              uplo_l,
                              m,
                              a + batch_idx * batch_stride_ev,
                              ew + batch_idx * batch_stride_ew,
                              compute_evs ? (ev + batch_idx * batch_stride_ev) : nullptr);
      }
    }
  }
};

template <>
struct SyevImplBody<VariantKind::GPU, Type::Code::FLOAT64> {
  void operator()(bool uplo_l,
                  int64_t m,
                  int64_t num_batches,
                  int64_t batch_stride_ew,
                  int64_t batch_stride_ev,
                  const double* a,
                  double* ew,
                  double* ev)
  {
    bool compute_evs = ev != nullptr;

    if (num_batches > 1 && get_cusolver_extra_symbols()->has_syev_batched) {
      syev_batched_template<double>(
        HIP_R_64F, HIP_R_64F, uplo_l, m, a, ew, compute_evs ? ev : nullptr, num_batches);
    } else {
      for (int64_t batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        syevd_template<double>(HIP_R_64F,
                               HIP_R_64F,
                               uplo_l,
                               m,
                               a + batch_idx * batch_stride_ev,
                               ew + batch_idx * batch_stride_ew,
                               compute_evs ? (ev + batch_idx * batch_stride_ev) : nullptr);
      }
    }
  }
};

template <>
struct SyevImplBody<VariantKind::GPU, Type::Code::COMPLEX64> {
  void operator()(bool uplo_l,
                  int64_t m,
                  int64_t num_batches,
                  int64_t batch_stride_ew,
                  int64_t batch_stride_ev,
                  const complex<float>* a,
                  float* ew,
                  complex<float>* ev)
  {
    bool compute_evs = ev != nullptr;

    if (num_batches > 1 && get_cusolver_extra_symbols()->has_syev_batched) {
      syev_batched_template<complex<float>>(
        HIP_R_32F,
        HIP_C_32F,
        uplo_l,
        m,
        reinterpret_cast<const hipComplex*>(a),
        ew,
        compute_evs ? reinterpret_cast<hipComplex*>(ev) : nullptr,
        num_batches);
    } else {
      for (int64_t batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        syevd_template<complex<float>>(
          HIP_R_32F,
          HIP_C_32F,
          uplo_l,
          m,
          reinterpret_cast<const hipComplex*>(a + batch_idx * batch_stride_ev),
          ew + batch_idx * batch_stride_ew,
          compute_evs ? reinterpret_cast<hipComplex*>(ev + batch_idx * batch_stride_ev) : nullptr);
      }
    }
  }
};

template <>
struct SyevImplBody<VariantKind::GPU, Type::Code::COMPLEX128> {
  void operator()(bool uplo_l,
                  int64_t m,
                  int64_t num_batches,
                  int64_t batch_stride_ew,
                  int64_t batch_stride_ev,
                  const complex<double>* a,
                  double* ew,
                  complex<double>* ev)
  {
    bool compute_evs = ev != nullptr;

    if (num_batches > 1 && get_cusolver_extra_symbols()->has_syev_batched) {
      syev_batched_template<complex<double>>(
        HIP_R_64F,
        HIP_C_64F,
        uplo_l,
        m,
        reinterpret_cast<const hipDoubleComplex*>(a),
        ew,
        compute_evs ? reinterpret_cast<hipDoubleComplex*>(ev) : nullptr,
        num_batches);
    } else {
      for (int64_t batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        syevd_template<complex<double>>(
          HIP_R_64F,
          HIP_C_64F,
          uplo_l,
          m,
          reinterpret_cast<const hipDoubleComplex*>(a + batch_idx * batch_stride_ev),
          ew + batch_idx * batch_stride_ew,
          compute_evs ? reinterpret_cast<hipDoubleComplex*>(ev + batch_idx * batch_stride_ev)
                      : nullptr);
      }
    }
  }
};

/*static*/ void SyevTask::gpu_variant(TaskContext context)
{
  syev_template<VariantKind::GPU>(context);
}

}  // namespace cupynumeric
