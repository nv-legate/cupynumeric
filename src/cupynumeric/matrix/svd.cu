#include "hip/hip_runtime.h"
/* Copyright 2024 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cupynumeric/matrix/svd.h"
#include "cupynumeric/matrix/svd_template.inl"

#include "cupynumeric/cuda_help.h"
#include <vector>
namespace cupynumeric {

using namespace legate;

template <typename VAL, typename DataType>
static inline void svd_template(DataType valTypeC,
                                DataType valTypeR,
                                int64_t m,
                                int64_t n,
                                int64_t k,
                                bool full_matrices,
                                const void* a,
                                void* u,
                                void* s,
                                void* vh,
                                hipStream_t stream)
{
  auto handle = get_cusolver();

  auto a_copy = create_buffer<VAL>(m * n, Memory::Kind::GPU_FB_MEM);
  CUPYNUMERIC_CHECK_CUDA(
    hipMemcpyAsync(a_copy.ptr(0), a, m * n * sizeof(VAL), hipMemcpyDeviceToDevice, stream));

  // a[m][n], u[m][m] s[k] vh[n][n]
  CHECK_CUSOLVER(hipsolverSetStream(handle, stream));

  size_t lwork_device, lwork_host;
  CHECK_CUSOLVER(cusolverDnXgesvd_bufferSize(handle,
                                             nullptr,
                                             full_matrices ? 'A' : 'S',
                                             'A',
                                             m,
                                             n,
                                             valTypeC,
                                             reinterpret_cast<void*>(a_copy.ptr(0)),
                                             m,
                                             valTypeR,
                                             s,
                                             valTypeC,
                                             u,
                                             m,
                                             valTypeC,
                                             vh,
                                             n,
                                             valTypeC,
                                             &lwork_device,
                                             &lwork_host));

  auto buffer = create_buffer<char>(lwork_device, Memory::Kind::GPU_FB_MEM);
  std::vector<char> buffer_host(std::max(1ul, lwork_host));
  auto info = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  CHECK_CUSOLVER(cusolverDnXgesvd(handle,
                                  nullptr,
                                  full_matrices ? 'A' : 'S',
                                  'A',
                                  m,
                                  n,
                                  valTypeC,
                                  reinterpret_cast<void*>(a_copy.ptr(0)),
                                  m,
                                  valTypeR,
                                  s,
                                  valTypeC,
                                  u,
                                  m,
                                  valTypeC,
                                  vh,
                                  n,
                                  valTypeC,
                                  buffer.ptr(0),
                                  lwork_device,
                                  buffer_host.data(),
                                  lwork_host,
                                  info.ptr(0)));

  CUPYNUMERIC_CHECK_CUDA(hipStreamSynchronize(stream));

  if (info[0] != 0) {
    throw legate::TaskException(SvdTask::ERROR_MESSAGE);
  }

  CUPYNUMERIC_CHECK_CUDA_STREAM(stream);

#ifdef DEBUG_CUPYNUMERIC
  assert(info[0] == 0);
#endif
}

template <>
struct SvdImplBody<VariantKind::GPU, Type::Code::FLOAT32> {
  TaskContext context;
  explicit SvdImplBody(TaskContext context) : context(context) {}

  void operator()(int64_t m,
                  int64_t n,
                  int64_t k,
                  bool full_matrices,
                  const float* a,
                  float* u,
                  float* s,
                  float* vh)
  {
    auto stream = context.get_task_stream();
    svd_template<float>(HIP_R_32F, HIP_R_32F, m, n, k, full_matrices, a, u, s, vh, stream);
  }
};

template <>
struct SvdImplBody<VariantKind::GPU, Type::Code::FLOAT64> {
  TaskContext context;
  explicit SvdImplBody(TaskContext context) : context(context) {}

  void operator()(int64_t m,
                  int64_t n,
                  int64_t k,
                  bool full_matrices,
                  const double* a,
                  double* u,
                  double* s,
                  double* vh)
  {
    auto stream = context.get_task_stream();
    svd_template<double>(HIP_R_64F, HIP_R_64F, m, n, k, full_matrices, a, u, s, vh, stream);
  }
};

template <>
struct SvdImplBody<VariantKind::GPU, Type::Code::COMPLEX64> {
  TaskContext context;
  explicit SvdImplBody(TaskContext context) : context(context) {}

  void operator()(int64_t m,
                  int64_t n,
                  int64_t k,
                  bool full_matrices,
                  const legate::Complex<float>* a,
                  legate::Complex<float>* u,
                  float* s,
                  legate::Complex<float>* vh)
  {
    auto stream = context.get_task_stream();
    svd_template<legate::Complex<float>>(HIP_C_32F,
                                         HIP_R_32F,
                                         m,
                                         n,
                                         k,
                                         full_matrices,
                                         reinterpret_cast<const hipComplex*>(a),
                                         reinterpret_cast<hipComplex*>(u),
                                         s,
                                         reinterpret_cast<hipComplex*>(vh),
                                         stream);
  }
};

template <>
struct SvdImplBody<VariantKind::GPU, Type::Code::COMPLEX128> {
  TaskContext context;
  explicit SvdImplBody(TaskContext context) : context(context) {}

  void operator()(int64_t m,
                  int64_t n,
                  int64_t k,
                  bool full_matrices,
                  const legate::Complex<double>* a,
                  legate::Complex<double>* u,
                  double* s,
                  legate::Complex<double>* vh)
  {
    auto stream = context.get_task_stream();
    svd_template<legate::Complex<double>>(HIP_C_64F,
                                          HIP_R_64F,
                                          m,
                                          n,
                                          k,
                                          full_matrices,
                                          reinterpret_cast<const hipDoubleComplex*>(a),
                                          reinterpret_cast<hipDoubleComplex*>(u),
                                          s,
                                          reinterpret_cast<hipDoubleComplex*>(vh),
                                          stream);
  }
};

/*static*/ void SvdTask::gpu_variant(TaskContext context)
{
  svd_template<VariantKind::GPU>(context);
}

}  // namespace cupynumeric
