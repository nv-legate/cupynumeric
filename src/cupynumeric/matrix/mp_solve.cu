#include "hip/hip_runtime.h"
/* Copyright 2024 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cupynumeric/matrix/mp_solve.h"
#include "cupynumeric/matrix/mp_solve_template.inl"

#include "cupynumeric/cuda_help.h"

namespace cupynumeric {

using namespace Legion;
using namespace legate;

template <typename VAL>
static inline void mp_solve_template(cal_comm_t comm,
                                     int nprow,
                                     int npcol,
                                     int64_t n,
                                     int64_t nrhs,
                                     int64_t nb,
                                     VAL* a_array,
                                     int64_t llda,
                                     VAL* b_array,
                                     int64_t lldb,
                                     hipStream_t ctx_stream)
{
  const auto trans = HIPBLAS_OP_N;

  auto handle = get_cusolvermp(ctx_stream);

  // synchronize all previous copies on default stream
  // cusolverMP has its unmodifiable stream to continue with
  CUPYNUMERIC_CHECK_CUDA(hipStreamSynchronize(ctx_stream));

  hipStream_t stream;
  CHECK_CUSOLVER(cusolverMpGetStream(handle, &stream));

  cusolverMpGrid_t grid = nullptr;
  CHECK_CUSOLVER(cusolverMpCreateDeviceGrid(
    handle, &grid, comm, nprow, npcol, CUSOLVERMP_GRID_MAPPING_COL_MAJOR));

  cusolverMpMatrixDescriptor_t a_desc = nullptr;
  CHECK_CUSOLVER(cusolverMpCreateMatrixDesc(
    &a_desc, grid, cudaTypeToDataType<VAL>::type, n, n, nb, nb, 0, 0, llda));

  cusolverMpMatrixDescriptor_t b_desc = nullptr;
  CHECK_CUSOLVER(cusolverMpCreateMatrixDesc(
    &b_desc, grid, cudaTypeToDataType<VAL>::type, n, nrhs, nb, nb, 0, 0, lldb));

  size_t getrf_device_buffer_size = 0;
  size_t getrf_host_buffer_size   = 0;
  CHECK_CUSOLVER(cusolverMpGetrf_bufferSize(handle,
                                            n,
                                            n,
                                            a_array,
                                            1,
                                            1,
                                            a_desc,
                                            nullptr,
                                            cudaTypeToDataType<VAL>::type,
                                            &getrf_device_buffer_size,
                                            &getrf_host_buffer_size));

  size_t getrs_device_buffer_size = 0;
  size_t getrs_host_buffer_size   = 0;
  CHECK_CUSOLVER(cusolverMpGetrs_bufferSize(handle,
                                            trans,
                                            n,
                                            nrhs,
                                            a_array,
                                            1,
                                            1,
                                            a_desc,
                                            nullptr,
                                            b_array,
                                            1,
                                            1,
                                            b_desc,
                                            cudaTypeToDataType<VAL>::type,
                                            &getrs_device_buffer_size,
                                            &getrs_host_buffer_size));

  // ensure non-empty buffers
  size_t device_buffer_size =
    std::max(std::max(getrf_device_buffer_size, getrs_device_buffer_size), 1ul);
  size_t host_buffer_size = std::max(std::max(getrf_host_buffer_size, getrs_host_buffer_size), 1ul);

  auto device_buffer = create_buffer<int8_t>(device_buffer_size, Memory::Kind::GPU_FB_MEM);
  auto host_buffer   = create_buffer<int8_t>(host_buffer_size, Memory::Kind::Z_COPY_MEM);
  auto info          = create_buffer<int32_t>(1, Memory::Kind::Z_COPY_MEM);

  // initialize to zero
  info[0] = 0;

  CHECK_CUSOLVER(cusolverMpGetrf(handle,
                                 n,
                                 n,
                                 a_array,
                                 1,
                                 1,
                                 a_desc,
                                 nullptr,
                                 cudaTypeToDataType<VAL>::type,
                                 device_buffer.ptr(0),
                                 device_buffer_size,
                                 host_buffer.ptr(0),
                                 host_buffer_size,
                                 info.ptr(0)));

  if (info[0] != 0) {
    throw legate::TaskException("Matrix is singular");
  }

  CHECK_CUSOLVER(cusolverMpGetrs(handle,
                                 trans,
                                 n,
                                 nrhs,
                                 a_array,
                                 1,
                                 1,
                                 a_desc,
                                 nullptr,
                                 b_array,
                                 1,
                                 1,
                                 b_desc,
                                 cudaTypeToDataType<VAL>::type,
                                 device_buffer.ptr(0),
                                 device_buffer_size,
                                 host_buffer.ptr(0),
                                 host_buffer_size,
                                 info.ptr(0)));

  // TODO: We need a deferred exception to avoid this synchronization
  CHECK_CAL(cal_stream_sync(comm, stream));
  CUPYNUMERIC_CHECK_CUDA_STREAM(stream);

  CHECK_CUSOLVER(cusolverMpDestroyMatrixDesc(a_desc));
  CHECK_CUSOLVER(cusolverMpDestroyMatrixDesc(b_desc));
  CHECK_CUSOLVER(cusolverMpDestroyGrid(grid));

  // FIXME: this should be synchronized with all participating tasks in order to quit gracefully
  if (info[0] != 0) {
    throw legate::TaskException("Matrix is singular");
  }
}

template <>
struct MpSolveImplBody<VariantKind::GPU, Type::Code::FLOAT32> {
  TaskContext context;
  explicit MpSolveImplBody(TaskContext context) : context(context) {}

  void operator()(cal_comm_t comm,
                  int nprow,
                  int npcol,
                  int64_t n,
                  int64_t nrhs,
                  int64_t nb,
                  float* a_array,
                  int64_t llda,
                  float* b_array,
                  int64_t lldb)
  {
    auto stream = context.get_task_stream();
    mp_solve_template(comm, nprow, npcol, n, nrhs, nb, a_array, llda, b_array, lldb, stream);
  }
};

template <>
struct MpSolveImplBody<VariantKind::GPU, Type::Code::FLOAT64> {
  TaskContext context;
  explicit MpSolveImplBody(TaskContext context) : context(context) {}

  void operator()(cal_comm_t comm,
                  int nprow,
                  int npcol,
                  int64_t n,
                  int64_t nrhs,
                  int64_t nb,
                  double* a_array,
                  int64_t llda,
                  double* b_array,
                  int64_t lldb)
  {
    auto stream = context.get_task_stream();
    mp_solve_template(comm, nprow, npcol, n, nrhs, nb, a_array, llda, b_array, lldb, stream);
  }
};

template <>
struct MpSolveImplBody<VariantKind::GPU, Type::Code::COMPLEX64> {
  TaskContext context;
  explicit MpSolveImplBody(TaskContext context) : context(context) {}

  void operator()(cal_comm_t comm,
                  int nprow,
                  int npcol,
                  int64_t n,
                  int64_t nrhs,
                  int64_t nb,
                  complex<float>* a_array,
                  int64_t llda,
                  complex<float>* b_array,
                  int64_t lldb)
  {
    auto stream = context.get_task_stream();
    mp_solve_template(comm,
                      nprow,
                      npcol,
                      n,
                      nrhs,
                      nb,
                      reinterpret_cast<hipComplex*>(a_array),
                      llda,
                      reinterpret_cast<hipComplex*>(b_array),
                      lldb,
                      stream);
  }
};

template <>
struct MpSolveImplBody<VariantKind::GPU, Type::Code::COMPLEX128> {
  TaskContext context;
  explicit MpSolveImplBody(TaskContext context) : context(context) {}

  void operator()(cal_comm_t comm,
                  int nprow,
                  int npcol,
                  int64_t n,
                  int64_t nrhs,
                  int64_t nb,
                  complex<double>* a_array,
                  int64_t llda,
                  complex<double>* b_array,
                  int64_t lldb)
  {
    auto stream = context.get_task_stream();
    mp_solve_template(comm,
                      nprow,
                      npcol,
                      n,
                      nrhs,
                      nb,
                      reinterpret_cast<hipDoubleComplex*>(a_array),
                      llda,
                      reinterpret_cast<hipDoubleComplex*>(b_array),
                      lldb,
                      stream);
  }
};

/*static*/ void MpSolveTask::gpu_variant(TaskContext context)
{
  mp_solve_template<VariantKind::GPU>(context);
}

namespace  // unnamed
{
static const auto cupynumeric_reg_task_ = []() -> char {
  MpSolveTask::register_variants();
  return 0;
}();
}  // namespace

}  // namespace cupynumeric
